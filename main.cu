#include "hip/hip_runtime.h"
#include <iostream>
#include "FractalCreator.h"
#include <math.h>
#include <complex>

using namespace mandelbrot;

FractalCreator fractalCreator(M_WIDTH, M_HEIGHT);


using namespace std;

void inicializar(void){
    glClearColor (0.0, 0.0, 0.0, 0.0);
    //glShadeModel (GL_FLAT);
}


void display_mandelbrot(){
    glClear(GL_COLOR_BUFFER_BIT);
    glColor3f(1.0, 1.0, 1.0);
    glPointSize(1.0);
		fractalCreator.drawFractal();
    glutSwapBuffers();
}

void scene(int w, int h){
    glViewport (0, 0, (GLsizei) M_WIDTH, (GLsizei) M_HEIGHT);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
		gluOrtho2D(0, M_WIDTH, M_HEIGHT, 0);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}

void keyboard_mandelbrot(unsigned char key, int x_, int y_){
    switch (key) {
        case 'q':
            exit(0);
            break;

        default:
            break;
    }
}

void mouseClick(int button, int state, int x, int y){

  if (button == GLUT_LEFT_BUTTON) {
    if (state == GLUT_DOWN) {
      fractalCreator.addZoom(x, M_HEIGHT - y, 0.1);
      fractalCreator.run();
      glutPostRedisplay();
    }
  }

}


int main(int argc, char** argv) {


	//FractalCreator fractalCreator(M_WIDTH, M_HEIGHT);

	fractalCreator.addColorRange(0.0, RGB(0,0,0));
  fractalCreator.addColorRange(0.2, RGB(255,0,0));
  fractalCreator.addColorRange(0.4, RGB(255,0,255));
  fractalCreator.addColorRange(0.6, RGB(0,255,255));
  fractalCreator.addColorRange(0.8, RGB(255,255,0));
  fractalCreator.addColorRange(1.0, RGB(255,255,255));


	//fractalCreator.addZoom(313, 330, 0.1);
  //fractalCreator.addZoom(825, 130, 0.1);
  fractalCreator.run();
  //fractalCreator.drawFractal();

/**/
  int max = 0;
  int maxi_c = 0;
  for (int i = 0; i < M_WIDTH * M_HEIGHT; i++) {
    if (fractalCreator.m_fractal[i] > max) {
      max = fractalCreator.m_fractal[i];
      maxi_c = 0;
    } else if (fractalCreator.m_fractal[i] == max){
      maxi_c++;
    }
  }
  int tot = 0;
  int max_histo = 0;
  int max_histo_rep = 0;
  for (int i = 0; i < Mandelbrot::MAX_ITERATIONS + 1; i++) {
    if (fractalCreator.m_histogram[i] > max_histo) {
      max_histo = fractalCreator.m_histogram[i];
      max_histo_rep = 1;
    } else if (fractalCreator.m_histogram[i] == max){
      max_histo_rep++;
    }
    tot += fractalCreator.m_histogram[i];
  }

  printf("maximo %d %d, max %d\n", max,fractalCreator.m_histogram[max], maxi_c);
  printf("Total: %d    Mas rep: %d    rep max: %d\n",tot, max_histo, max_histo_rep );
/**/



	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	glutInitWindowSize(M_WIDTH, M_HEIGHT);
	glutInitWindowPosition(100, 100);
	glutCreateWindow("MANDELBROT SET");
	inicializar();

	glutKeyboardFunc(keyboard_mandelbrot);
  glutMouseFunc(mouseClick);
	glutDisplayFunc(display_mandelbrot);
	glutReshapeFunc(scene);
	glutMainLoop();

	printf("Finished.\n");
	return 0;
}
